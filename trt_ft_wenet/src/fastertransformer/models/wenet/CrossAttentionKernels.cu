#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/fastertransformer/kernels/bfloat16_fallback_kenrels.cuh"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/models/wenet/WenetKernels.h"
#include "src/fastertransformer/utils/cuda_utils.h"

namespace fastertransformer {

template<typename T>
__global__ void addQKVBiasTranspose(T* q_out,
                                    T* k_out,
                                    T* v_out,
                                    const T* __restrict q_in,
                                    const T* __restrict bias_q,
                                    const T* __restrict k_in,
                                    const T* __restrict bias_k,
                                    const T* __restrict v_in,
                                    const T* __restrict bias_v,
                                    const int batch_size,
                                    const int seq_len1,
                                    const int seq_len2,
                                    const int head_num,
                                    const int size_per_head)
{
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id1 = batch_id * seq_len1 + word_id;
    const int row_id2 = batch_id * seq_len2 + word_id;

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x) {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id1 = batch_id * (head_num * seq_len1 * size_per_head) + head_id * seq_len1 * size_per_head
                               + word_id * size_per_head + size_id;
        const int target_id2 = batch_id * (head_num * seq_len2 * size_per_head) + head_id * seq_len2 * size_per_head
                               + word_id * size_per_head + size_id;

        const int src_id1 = row_id1 * n + col_id;
        const int src_id2 = row_id2 * n + col_id;

        if (word_id < seq_len1)
            q_out[target_id1] = __ldg(&q_in[src_id1]) + __ldg(&bias_q[col_id]);

        if (word_id < seq_len2) {
            k_out[target_id2] = __ldg(&k_in[src_id2]) + __ldg(&bias_k[col_id]);

            v_out[target_id2] = __ldg(&v_in[src_id2]) + __ldg(&bias_v[col_id]);
        }
    }
}

template<typename T>
__global__ void QKVTranspose(T* q_out,
                             T* k_out,
                             T* v_out,
                             const T* __restrict q_in,
                             const T* __restrict k_in,
                             const T* __restrict v_in,
                             const int batch_size,
                             const int seq_len1,
                             const int seq_len2,
                             const int head_num,
                             const int size_per_head)
{
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id1 = batch_id * seq_len1 + word_id;
    const int row_id2 = batch_id * seq_len2 + word_id;

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x) {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id1 = batch_id * (head_num * seq_len1 * size_per_head) + head_id * seq_len1 * size_per_head
                               + word_id * size_per_head + size_id;
        const int target_id2 = batch_id * (head_num * seq_len2 * size_per_head) + head_id * seq_len2 * size_per_head
                               + word_id * size_per_head + size_id;

        const int src_id1 = row_id1 * n + col_id;
        const int src_id2 = row_id2 * n + col_id;

        if (word_id < seq_len1)
            q_out[target_id1] = __ldg(&q_in[src_id1]);

        if (word_id < seq_len2) {
            k_out[target_id2] = __ldg(&k_in[src_id2]);
            v_out[target_id2] = __ldg(&v_in[src_id2]);
        }
    }
}

template<typename T>
void invokeAddQKVBiasTranspose(T* q_buf,
                               T* k_buf,
                               T* v_buf,
                               T* Q,
                               const T* bias_Q,
                               T* K,
                               const T* bias_K,
                               T* V,
                               const T* bias_V,
                               const int batch_size,
                               const int seq_len1,
                               const int seq_len2,
                               const int head_num,
                               const int size_per_head,
                               hipStream_t stream)
{
    const int k = head_num * size_per_head;
    int seq_len12 = max(seq_len1, seq_len2);
    dim3 grid(batch_size, seq_len12);
    bool is_add_bias = bias_Q != nullptr;
    if (sizeof(T) == 4 || k % 2 != 0) {
        dim3 block(min(k, 512));
        if (is_add_bias) {
            addQKVBiasTranspose<T><<<grid, block, 0, stream>>>(q_buf,
                                                               k_buf,
                                                               v_buf,
                                                               Q,
                                                               bias_Q,
                                                               K,
                                                               bias_K,
                                                               V,
                                                               bias_V,
                                                               batch_size,
                                                               seq_len1,
                                                               seq_len2,
                                                               head_num,
                                                               size_per_head);
        }
        else {
            QKVTranspose<T><<<grid, block, 0, stream>>>(
                q_buf, k_buf, v_buf, Q, K, V, batch_size, seq_len1, seq_len2, head_num, size_per_head);
        }
        sync_check_cuda_error();
    }
    else {
        dim3 block(min(k / 2, 512));
        if (is_add_bias) {
            addQKVBiasTranspose<half2><<<grid, block, 0, stream>>>((half2*)q_buf,
                                                                   (half2*)k_buf,
                                                                   (half2*)v_buf,
                                                                   (const half2*)Q,
                                                                   (const half2*)bias_Q,
                                                                   (const half2*)K,
                                                                   (const half2*)bias_K,
                                                                   (const half2*)V,
                                                                   (const half2*)bias_V,
                                                                   batch_size,
                                                                   seq_len1,
                                                                   seq_len2,
                                                                   head_num,
                                                                   size_per_head / 2);
        }
        else {
            QKVTranspose<half2><<<grid, block, 0, stream>>>((half2*)q_buf,
                                                            (half2*)k_buf,
                                                            (half2*)v_buf,
                                                            (const half2*)Q,
                                                            (const half2*)K,
                                                            (const half2*)V,
                                                            batch_size,
                                                            seq_len1,
                                                            seq_len2,
                                                            head_num,
                                                            size_per_head / 2);
        }
        sync_check_cuda_error();
    }
}

template void invokeAddQKVBiasTranspose(float* q_buf,
                                        float* k_buf,
                                        float* v_buf,
                                        float* Q,
                                        const float* bias_Q,
                                        float* K,
                                        const float* bias_K,
                                        float* V,
                                        const float* bias_V,
                                        const int batch_size,
                                        const int seq_len1,
                                        const int seq_len2,
                                        const int head_num,
                                        const int size_per_head,
                                        hipStream_t stream);

template void invokeAddQKVBiasTranspose(half* q_buf,
                                        half* k_buf,
                                        half* v_buf,
                                        half* Q,
                                        const half* bias_Q,
                                        half* K,
                                        const half* bias_K,
                                        half* V,
                                        const half* bias_V,
                                        const int batch_size,
                                        const int seq_len1,
                                        const int seq_len2,
                                        const int head_num,
                                        const int size_per_head,
                                        hipStream_t stream);

// TODO(bhsueh) Rename the softmax_kernel_v4 to softmax_kernel
template<int ITEMS_PER_THREAD, typename T, typename T_IN>
__global__ void softmax_kernel_v4(T* qk_buf_,
                                  const T_IN* qk_buf_src,
                                  const T* attr_mask,
                                  const int batch_size,
                                  const int head_num,
                                  const int seq_len1,
                                  const int seq_len2,
                                  const T scalar)
{
    for (int seq_id = blockIdx.x; seq_id < seq_len1; seq_id += gridDim.x) {
        float data[ITEMS_PER_THREAD];
        int qk_offset;
        __shared__ float s_mean, s_max;
        float local_max = -1e20f;
        for (int i = 0; blockDim.x * i + threadIdx.x < seq_len2; i++) {
            qk_offset =
                ((blockIdx.y * head_num + blockIdx.z) * seq_len1 + seq_id) * seq_len2 + blockDim.x * i + threadIdx.x;
            int mask_offset = (blockIdx.y * seq_len1 + seq_id) * seq_len2 + blockDim.x * i + threadIdx.x;

            float qk = static_cast<float>(qk_buf_src[qk_offset]);

            float mask_val = static_cast<float>(ldg(&attr_mask[mask_offset]));

            mask_val = (1.0f - mask_val) * -10000.0f;

            data[i] = qk * static_cast<float>(scalar) + mask_val;
            local_max = fmax(local_max, data[i]);
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int i = 0; blockDim.x * i + threadIdx.x < seq_len2; i++) {
            data[i] = __expf(data[i] - s_max);
            local_sum += data[i];
        }
        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);
        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < seq_len2; i++) {
            qk_offset =
                ((blockIdx.y * head_num + blockIdx.z) * seq_len1 + seq_id) * seq_len2 + blockDim.x * i + threadIdx.x;
            qk_buf_[qk_offset] = (T)(data[i] * s_mean);
        }
    }
}

template<typename T, int ITEMS_PER_THREAD>
__global__ void softmax_kernel_v4_half2(T* qk_buf_,
                                        const T* attr_mask,
                                        const int batch_size,
                                        const int head_num,
                                        const int seq_len1,
                                        const int seq_len2,
                                        const T scalar)
{
    using T2 = typename TypeConverter<T>::Type;
    T2* qk_buf_half2 = (T2*)qk_buf_;
    const T2* attr_mask_half2 = (const T2*)attr_mask;

    for (int seq_id = blockIdx.x; seq_id < seq_len1; seq_id += gridDim.x) {
        T2 data[ITEMS_PER_THREAD];
        int qk_offset;
        __shared__ float s_mean, s_max;
        float local_max = -1e20f;
        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len2 / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((blockIdx.y * head_num + blockIdx.z) * seq_len1 + seq_id) * (seq_len2 / 2) + blockDim.x * i
                        + threadIdx.x;
            int mask_offset = (blockIdx.y * seq_len1 + seq_id) * (seq_len2 / 2) + blockDim.x * i + threadIdx.x;

            T2 qk = qk_buf_half2[qk_offset];
            T2 mask_val = ldg(&attr_mask_half2[mask_offset]);
            mask_val = hmul2<T2>(hsub2<T2>(float2type2<T2>(1.0f), mask_val), float2type2<T2>(-10000.0f));

            data[i] = hadd2<T2>(hmul2<T2>(qk, type2type2<T, T2>(scalar)), mask_val);

            local_max = fmax(local_max, fmax((float)data[i].x, (float)data[i].y));
        }

        float max_val = blockDim.x <= 32 ? warpReduceMax(local_max) : blockReduceMax<float>(local_max);
        if (threadIdx.x == 0) {
            s_max = max_val;
        }
        __syncthreads();

        float local_sum = 0;
        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len2 / 2) && i < ITEMS_PER_THREAD; i++) {
            data[i] = hexp2<T2>(hsub2<T2>(data[i], float2type2<T2>(s_max)));
            local_sum += (float)(data[i].x + data[i].y);
        }

        float sum_val = blockDim.x <= 32 ? warpReduceSum(local_sum) : blockReduceSum<float>(local_sum);

        if (threadIdx.x == 0) {
            s_mean = sum_val + 1e-6f;
            s_mean = __fdividef(1.0f, s_mean);
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len2 / 2) && i < ITEMS_PER_THREAD; i++) {
            qk_offset = ((blockIdx.y * head_num + blockIdx.z) * seq_len1 + seq_id) * (seq_len2 / 2) + blockDim.x * i
                        + threadIdx.x;
            qk_buf_half2[qk_offset] = hmul2<T2>(data[i], float2type2<T2>(s_mean));
        }
    }
}

template<typename T, int ITEMS_PER_THREAD, int NUM>
__global__ void softmax_kernel_v5_half2(T* qk_buf_,
                                        const T* attr_mask,
                                        const int batch_size,
                                        const int head_num,
                                        const int seq_len1,
                                        const int seq_len2,
                                        const T scalar)
{
    using T2 = typename TypeConverter<T>::Type;
    T2* qk_buf_half2 = (T2*)qk_buf_;

    const T2* attr_mask_half2 = (const T2*)attr_mask;

    for (int seq_id = blockIdx.x; seq_id < seq_len1; seq_id += gridDim.x * NUM) {
        T2 data[NUM][ITEMS_PER_THREAD];

        int qk_offset[NUM];

        __shared__ float s_sum[NUM], s_max[NUM];
        float local_max[NUM];
#pragma unroll
        for (int j = 0; j < NUM; j++) {
            local_max[j] = -1e20f;
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len2 / 2) && i < ITEMS_PER_THREAD; i++) {
            int mask_offset[NUM];
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk_offset[j] =
                    ((blockIdx.y * head_num + blockIdx.z) * seq_len1 + seq_id + j * gridDim.x) * (seq_len2 / 2)
                    + blockDim.x * i + threadIdx.x;
                mask_offset[j] =
                    (blockIdx.y * seq_len1 + seq_id + j * gridDim.x) * (seq_len2 / 2) + blockDim.x * i + threadIdx.x;
            }

            T2 mask_val[NUM];
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                mask_val[j] = ldg(&attr_mask_half2[mask_offset[j]]);
            }

            T2 qk[NUM];
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk[j] = qk_buf_half2[qk_offset[j]];
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                mask_val[j] = hmul2<T2>(hsub2<T2>(float2type2<T2>(1.0f), mask_val[j]), float2type2<T2>(-10000.0f));
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                data[j][i] = hadd2<T2>(hmul2<T2>(qk[j], type2type2<T, T2>(scalar)), mask_val[j]);
                local_max[j] = fmax(local_max[j], fmax((float)data[j][i].x, (float)data[j][i].y));
            }
        }

        if (blockDim.x <= 32) {
            warpReduceMaxV2<float, NUM>(local_max);
        }
        else {
            blockReduceMaxV2<float, NUM>(local_max);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                s_max[j] = local_max[j];
            }
        }
        __syncthreads();

        float local_sum[NUM];
#pragma unroll
        for (int j = 0; j < NUM; j++) {
            local_sum[j] = {0.f};
        }

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len2 / 2) && i < ITEMS_PER_THREAD; i++) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                data[j][i] = hexp2<T2>(hsub2<T2>(data[j][i], float2type2<T2>(s_max[j])));
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                local_sum[j] += (float)(data[j][i].x + data[j][i].y);
            }
        }

        if (blockDim.x <= 32) {
            warpReduceSumV2<float, NUM>(local_sum);
        }
        else {
            blockReduceSumV2<float, NUM>(local_sum);
        }

        if (threadIdx.x == 0) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                s_sum[j] = __fdividef(1.0f, local_sum[j] + 1e-6f);
            }
        }
        __syncthreads();

        for (int i = 0; blockDim.x * i + threadIdx.x < (seq_len2 / 2) && i < ITEMS_PER_THREAD; i++) {
#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk_offset[j] =
                    ((blockIdx.y * head_num + blockIdx.z) * seq_len1 + seq_id + j * gridDim.x) * (seq_len2 / 2)
                    + blockDim.x * i + threadIdx.x;
            }

#pragma unroll
            for (int j = 0; j < NUM; j++) {
                qk_buf_half2[qk_offset[j]] = hmul2<T2>(data[j][i], float2type2<T2>(s_sum[j]));
            }
        }
    }
}

#define SOFTMAX_KERNEL(ITEMS_PER_THREAD)                                                                               \
    block.x /= ITEMS_PER_THREAD;                                                                                       \
    assert(block.x <= 1024);                                                                                           \
    if (is_half2) {                                                                                                    \
        if (grid.x % 4 == 0) {                                                                                         \
            grid.x /= 4;                                                                                               \
            softmax_kernel_v5_half2<half, ITEMS_PER_THREAD, 4><<<grid, block, 0, stream>>>(                            \
                (half*)buffer, (const half*)attr_mask, batch_size, head_num, seq_len1, seq_len2, (const half)scalar);  \
        }                                                                                                              \
        else {                                                                                                         \
            softmax_kernel_v4_half2<half, ITEMS_PER_THREAD><<<grid, block, 0, stream>>>(                               \
                (half*)buffer, (const half*)attr_mask, batch_size, head_num, seq_len1, seq_len2, (const half)scalar);  \
        }                                                                                                              \
    }                                                                                                                  \
    else {                                                                                                             \
        softmax_kernel_v4<ITEMS_PER_THREAD, T, T_IN><<<grid, block, 0, stream>>>(                                      \
            buffer, buffer_src, attr_mask, batch_size, head_num, seq_len1, seq_len2, scalar);                          \
    }

#ifdef ENABLE_BF16
#define SOFTMAX_KERNEL_BF16(ITEMS_PER_THREAD)                                                                          \
    block.x /= ITEMS_PER_THREAD;                                                                                       \
    assert(block.x <= 1024);                                                                                           \
    if (is_half2) {                                                                                                    \
        if (grid.x % 4 == 0) {                                                                                         \
            grid.x /= 4;                                                                                               \
            softmax_kernel_v5_half2<__hip_bfloat16, ITEMS_PER_THREAD, 4>                                                \
                <<<grid, block, 0, stream>>>((__hip_bfloat16*)buffer,                                                   \
                                             (const __hip_bfloat16*)attr_mask,                                          \
                                             batch_size,                                                               \
                                             head_num,                                                                 \
                                             seq_len1,                                                                 \
                                             seq_len2,                                                                 \
                                             (const __hip_bfloat16)scalar);                                             \
        }                                                                                                              \
        else {                                                                                                         \
            softmax_kernel_v4_half2<__hip_bfloat16, ITEMS_PER_THREAD>                                                   \
                <<<grid, block, 0, stream>>>((__hip_bfloat16*)buffer,                                                   \
                                             (const __hip_bfloat16*)attr_mask,                                          \
                                             batch_size,                                                               \
                                             head_num,                                                                 \
                                             seq_len1,                                                                 \
                                             seq_len2,                                                                 \
                                             (const __hip_bfloat16)scalar);                                             \
        }                                                                                                              \
    }                                                                                                                  \
    else {                                                                                                             \
        softmax_kernel_v4<ITEMS_PER_THREAD, __hip_bfloat16, T_IN><<<grid, block, 0, stream>>>(                          \
            buffer, buffer_src, attr_mask, batch_size, head_num, seq_len1, seq_len2, scalar);                          \
    }
#endif  // ENABLE_BF16

template<typename T, typename T_IN>
void invokeMaskedSoftMax(T* buffer,
                         const T_IN* buffer_src,
                         const T* attr_mask,
                         const int batch_size,
                         const int seq_len1,
                         const int seq_len2,
                         const int head_num,
                         const T scalar,
                         hipStream_t stream)
{

    dim3 grid(seq_len1, batch_size, head_num);
    if (batch_size * head_num > 360) {
        grid.x = ceil(float(seq_len1) / 32.0f);
    }

    bool is_half2 = sizeof(T) == 2 && sizeof(T_IN) == 2 && seq_len2 % 2 == 0;
    dim3 block((seq_len2 / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 3072 && block.x <= 4096) {
        SOFTMAX_KERNEL(4)
    }
    if (block.x > 2048) {
        SOFTMAX_KERNEL(3)
    }
    else if (block.x > 1024) {
        SOFTMAX_KERNEL(2)
    }
    else if (block.x > 0) {
        SOFTMAX_KERNEL(1)
    }
    else {
        FT_CHECK(seq_len2 <= 4096);
    }
}

#ifdef ENABLE_BF16
template<>
void invokeMaskedSoftMax(__hip_bfloat16* buffer,
                         const __hip_bfloat16* buffer_src,
                         const __hip_bfloat16* attr_mask,
                         const int batch_size,
                         const int seq_len1,
                         const int seq_len2,
                         const int head_num,
                         const __hip_bfloat16 scalar,
                         hipStream_t stream)
{

    using T_IN = __hip_bfloat16;
    dim3 grid(seq_len1, batch_size, head_num);
    if (batch_size * head_num > 360) {
        grid.x = ceil(float(seq_len1) / 32.0f);
    }

    bool is_half2 = seq_len2 % 2 == 0;
    dim3 block((seq_len2 / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 3072 && block.x <= 4096) {
        SOFTMAX_KERNEL_BF16(4)
    }
    if (block.x > 2048) {
        SOFTMAX_KERNEL_BF16(3)
    }
    else if (block.x > 1024) {
        SOFTMAX_KERNEL_BF16(2)
    }
    else if (block.x > 0) {
        SOFTMAX_KERNEL_BF16(1)
    }
    else {
        FT_CHECK(seq_len2 <= 4096);
    }
}

template<>
void invokeMaskedSoftMax(__hip_bfloat16* buffer,
                         const float* buffer_src,
                         const __hip_bfloat16* attr_mask,
                         const int batch_size,
                         const int seq_len1,
                         const int seq_len2,
                         const int head_num,
                         const __hip_bfloat16 scalar,
                         hipStream_t stream)
{
    using T_IN = float;
    dim3 grid(seq_len1, batch_size, head_num);
    if (batch_size * head_num > 360) {
        grid.x = ceil(float(seq_len1) / 32.0f);
    }

    bool is_half2 = false;
    dim3 block((seq_len2 / (is_half2 ? 2 : 1) + 31) / 32 * 32);

    if (block.x > 3072 && block.x <= 4096) {
        SOFTMAX_KERNEL_BF16(4)
    }
    if (block.x > 2048) {
        SOFTMAX_KERNEL_BF16(3)
    }
    else if (block.x > 1024) {
        SOFTMAX_KERNEL_BF16(2)
    }
    else if (block.x > 0) {
        SOFTMAX_KERNEL_BF16(1)
    }
    else {
        FT_CHECK(seq_len2 <= 4096);
    }
}
#endif  // ENABLE_BF16

template void invokeMaskedSoftMax(float* buffer,
                                  const float* buffer_src,
                                  const float* attr_mask,
                                  const int batch_size,
                                  const int seq_len1,
                                  const int seq_len2,
                                  const int head_num,
                                  const float scalar,
                                  hipStream_t stream);

template void invokeMaskedSoftMax(half* buffer,
                                  const float* buffer_src,
                                  const half* attr_mask,
                                  const int batch_size,
                                  const int seq_len1,
                                  const int seq_len2,
                                  const int head_num,
                                  const half scalar,
                                  hipStream_t stream);

template void invokeMaskedSoftMax(half* buffer,
                                  const half* buffer_src,
                                  const half* attr_mask,
                                  const int batch_size,
                                  const int seq_len1,
                                  const int seq_len2,
                                  const int head_num,
                                  const half scalar,
                                  hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeMaskedSoftMax(__hip_bfloat16* buffer,
                                  const __hip_bfloat16* buffer_src,
                                  const __hip_bfloat16* attr_mask,
                                  const int batch_size,
                                  const int seq_len1,
                                  const int seq_len2,
                                  const int head_num,
                                  const __hip_bfloat16 scalar,
                                  hipStream_t stream);

template void invokeMaskedSoftMax(__hip_bfloat16* buffer,
                                  const float* buffer_src,
                                  const __hip_bfloat16* attr_mask,
                                  const int batch_size,
                                  const int seq_len1,
                                  const int seq_len2,
                                  const int head_num,
                                  const __hip_bfloat16 scalar,
                                  hipStream_t stream);
#endif  // ENABLE_BF16

}  // namespace fastertransformer
